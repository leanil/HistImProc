#include "hip/hip_runtime.h"
#include "kernel.h"
#include <hip/hip_runtime.h>
#include <>

#include <iostream>
#include <cstdio>

using namespace std;
using namespace cv;

const int BLOCK_SIZE = 16, GRID_SIZE = 8;
const dim3 dim_block(BLOCK_SIZE, BLOCK_SIZE), dim_grid(GRID_SIZE, GRID_SIZE);
const int stride = 16;
template<class T>
inline T div_ceil(T a, T b) { return a / b + !(a % b); }

#ifdef _DEBUG
#define check(code) checkCuda(code, __FILE__, __LINE__)
#else
#define check(code) code
#endif

inline void checkCuda(hipError_t code, const char *file, int line, bool abort = true) {
	if (code != hipSuccess) {
		cerr << "Cuda error: " << hipGetErrorString(code) << " " << file << " " << line << endl;
		if (abort) { exit(code); }
	}
}

void copy_image_to_device(const Mat& img, uchar*& d_img, size_t& pitch) {
	const unsigned &width = img.size().width, &height = img.size().height;
	check(hipMallocPitch(&d_img, &pitch, width * img.elemSize(), height));
	check(hipMemcpy2D(d_img, pitch, img.data, width, width * img.elemSize(), height, hipMemcpyHostToDevice));
}

__global__ void brightness_kernel(uchar* img, size_t pitch, int width, int height, int diff) {
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int nx = blockDim.x * gridDim.x;
	int ny = blockDim.y * gridDim.y;
	for (int y = row; y < height; y += ny) {
		for (int x = col; x < width; x += nx) {
			uchar* pos = img + y*pitch + x;
			if (diff + *pos < 0) { *pos = 0; }
			else if (diff + *pos > 255) { *pos = 255; }
			else { *pos += diff; }
		}
	}
}

__global__ void histogram_kernel(uchar* img, size_t pitch, int width, int height, float* hist) {
	__shared__ float loc_hist[256];
	int col = (blockIdx.x * blockDim.x + threadIdx.x) * stride;
	int row = (blockIdx.y * blockDim.y + threadIdx.y) * stride;
	int nx = blockDim.x * gridDim.x * stride;
	int ny = blockDim.y * gridDim.y * stride;

	int linear_idx = threadIdx.y * blockDim.x + threadIdx.x;
	int block_size = blockDim.x * blockDim.y;
	for (int i = linear_idx; i < 256; i += block_size) {
		loc_hist[i] = 0;
	}

	for (int y = row; y < height; y += ny) {
		for (int x = col; x < width; x += nx) {
			for (int dy = 0; dy < stride; ++dy) {
				for (int dx = 0; dx < stride; ++dx) {
					uchar* pos = img + (y + dy)*pitch + x + dx;
					atomicAdd(loc_hist + *pos, 1.0f);
				}
			}
		}
	}
	__syncthreads();
	for (int i = linear_idx; i < 256; i += block_size) {
		atomicAdd(hist + i, loc_hist[i]);
	}
}

Mat adjust_brightness(const Mat& img, int diff) {
	uchar* d_img;
	size_t pitch;
	copy_image_to_device(img, d_img, pitch);

	const unsigned &width = img.size().width, &height = img.size().height;
	brightness_kernel << <dim_grid, dim_block >> >(d_img, pitch, width, height, diff);

	Mat result(height, width, img.type());
	check(hipMemcpy2D(result.data, width, d_img, pitch, width * img.elemSize(), height, hipMemcpyDeviceToHost));
	return result;
}

Mat calculate_histogram(const cv::Mat& img) {
	uchar* d_img;
	size_t pitch;
	copy_image_to_device(img, d_img, pitch);

	const unsigned &width = img.size().width, &height = img.size().height;
	float* d_hist;
	check(hipMalloc(&d_hist, 256 * sizeof(float)));
	check(hipMemset(d_hist, 0.0f, 256 * sizeof(float)));

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	histogram_kernel << <dim_grid, dim_block >> >(d_img, pitch, width, height, d_hist);
	hipEventRecord(stop);

	Mat result(256, 1, CV_32FC1);
	check(hipMemcpy(result.data, d_hist, result.total()*result.elemSize(), hipMemcpyDeviceToHost));
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	cout << "histogram calculation: " << milliseconds << " ms\n";
	return result;
}